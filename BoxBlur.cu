#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include "lodepng.h"

/*****************
 2050052 Jimmy Shrestha

command to run the code
//nvcc -o Blur Task4.cu lodepng.cpp
//./Blur
 *****************/



__global__ void ImageBlur(unsigned char * gpu_imgOutput, unsigned char * gpu_imgInput,unsigned int w,unsigned int h,unsigned int blur, unsigned int bluryMD){

//initialization 
	int red = 0;
	int green = 0;
	int blue = 0;
	int x,y;
	int C = 0;


	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int pixel = idx*4;

	for(x = (pixel - (4 * blur)); x <=  (pixel + (4 * blur)); x+=4){
	//the checking of the pixel which is found at x
		if ((x > 0) && x < (h * w * 4) && ((x-4)/(4*w) == pixel/(4*w))){
			for(y = (x - (16 * w * blur)); y <=  (x + (16 * w *blur)); y+=(4*w)){
				if(y > 0 && y < ((h * w * 4))){
					red += gpu_imgInput[y];
					green += gpu_imgInput[1+y];
					blue += gpu_imgInput[2+y]; 
					C++;
				}
			}
		}
	}

	gpu_imgOutput[pixel] = red / C;
	gpu_imgOutput[1+pixel] = green / C;
	gpu_imgOutput[2+pixel] = blue / C;
	gpu_imgOutput[3+pixel] = gpu_imgInput[3+pixel];
}

int main(int argc, char **argv){



	unsigned int bluryMD = 3;
	unsigned int blur = (bluryMD - 1) / 2; // the matrix dimension of the blur
	unsigned int error;
	unsigned int encryptError;
	unsigned char* img;
	unsigned int w;
	unsigned int h;
	const char* filename = "Image.png";  //the image which is given as input
	const char* newFileName = "blur Image.png";//the blur image which comes as a result 

	error = lodepng_decode32_file(&img, &w, &h, filename);
	if(error){
		printf("error %u: %s\n", error, lodepng_error_text(error));
	}

	const int ARRAY_SIZE = w*h*4;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);

	unsigned char host_imgInput[ARRAY_SIZE * 4];
	unsigned char host_imgOutput[ARRAY_SIZE * 4];

	for (int i = 0; i < ARRAY_SIZE; i++) {
		host_imgInput[i] = img[i];
	}

	// declaration of the variable
	unsigned char * d_in;
	unsigned char * d_out;
	

// allocation of the GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);


	hipMemcpy(d_in, host_imgInput, ARRAY_BYTES, hipMemcpyHostToDevice);


	// kernel function launching
	ImageBlur<<<h, w>>>(d_out, d_in, w, h, blur, bluryMD);

		//copy result back to cpu
	hipMemcpy(host_imgOutput, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	printf("the blur Image has been created!\n");
	
	encryptError = lodepng_encode32_file(newFileName, host_imgOutput, w, h);
	if(encryptError){
		printf("error occured %u: %s\n", error, lodepng_error_text(encryptError));
	}

//memory allocation free
	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
